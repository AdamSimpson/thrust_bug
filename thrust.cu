#include "hip/hip_runtime.h"
#include <thrust/execution_policy.h>
#include <thrust/version.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/remove.h>
#include <thrust/binary_search.h>
#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <iostream>
#include <thrust/system_error.h>
#include "hip/hip_runtime.h"

struct LessThanOrEqual {
  LessThanOrEqual(double min): x_min(min) {}

  double x_min;

  __host__ __device__
  bool operator()(const double x) {
    return (x <= x_min);
  }
};


// C wrapper function for thrust copy_if with LessThanOrEqual predicate
extern "C" void CopyIfLessThanOrEqual(double min,
                                      int *input,
                                      int input_count,
                                      double * stencil,
                                      int * output,
                                      int * output_count,
                                      void* hip_stream) {

  int *end_pointer = thrust::copy_if(thrust::cuda::par.on((hipStream_t)hip_stream),
                                     input,
                                     input + input_count,
                                     stencil,
                                     output,
                                     LessThanOrEqual(min));
  *output_count = end_pointer - output;
}
