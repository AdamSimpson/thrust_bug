#include "hip/hip_runtime.h"
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/remove.h>
#include <thrust/binary_search.h>
#include <thrust/device_ptr.h>
#include <iostream>
#include "hip/hip_runtime.h"

extern "C" void CopyIfLessThanOrEqual( double min,
                                   int *const input,
                                   int input_count,
                                   double * stencil,
                                   int * output,
                                   int * output_count,
                                   void* hip_stream);

/*
struct LessThanOrEqual {
  LessThanOrEqual(double min): x_min(min) {}

  double x_min;

  __host__ __device__
  bool operator()(const double x) {
    return (x <= x_min);
  }
};

void CopyIfLessThanOrEqual(const double min,
                           const int *const input,
                           const int input_count,
                           const double *const stencil,
                           int *const output,
                           int *const output_count,
                           const void *const hip_stream) {

  int *end_pointer = thrust::copy_if(thrust::cuda::par.on((hipStream_t)hip_stream),
                                     input,
                                     input + input_count,
                                     stencil,
                                     output,
                                     LessThanOrEqual(min));
  *output_count = end_pointer - output;
}
*/

int main(int argc, char **argv)
{
  int count = 1000000;

  thrust::host_vector<int> h_input(count);
  thrust::sequence(h_input.begin(), h_input.end());

  thrust::host_vector<double> h_stencil(count);
  thrust::fill(h_stencil.begin(), h_stencil.end() - count/2, 1.5);
  thrust::fill(h_stencil.begin()+count/2, h_stencil.end(), 2.0);

  thrust::host_vector<int> h_output(count);
  thrust::host_vector<int> h_output2(count);

  thrust::device_vector<int> d_input = h_input;
  thrust::device_vector<double> d_stencil =h_stencil;
  thrust::device_vector<int> d_output = h_output;
  thrust::device_vector<int> d_output2 = h_output2;
  int output_count, output_count2;

  hipStream_t stream;
  hipStreamCreate(&stream);

  CopyIfLessThanOrEqual(
                        1.75,
                        thrust::raw_pointer_cast(d_input.data()),
                        count,
                        thrust::raw_pointer_cast(d_stencil.data()),
                        thrust::raw_pointer_cast(d_output.data()),
                        &output_count,
                        stream);

  CopyIfLessThanOrEqual(
                        3.0,
                        thrust::raw_pointer_cast(d_input.data()),
                        count,
                        thrust::raw_pointer_cast(d_stencil.data()),
                        thrust::raw_pointer_cast(d_output2.data()),
                        &output_count2,
                        stream);

  std::cout<<"copied: "<<output_count<<", "<<output_count2<<std::endl;
}
